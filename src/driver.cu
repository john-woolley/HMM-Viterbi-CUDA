#include "hip/hip_runtime.h"
/**
 * Driver code to read serialised input data for the Viterbi algorithm.
 * Requires an implementation of the Viterbi algorithm to produce output.
 *
 * COMP90025 Parallel and Multicore Computing
 * Qifan Deng (qifand@student.unimelb.edu.au)
 * Alan Ung (alanu@student.unimelb.edu.au)
 * Semester 2, 2019
 */

#include "../headers/driver.h"
#include "../headers/viterbi_sequential.h"
#include "../headers/viterbi_cuda.h"
#include <sys/time.h>

u_int64_t GetTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * (u_int64_t) 1000000 + tv.tv_usec;
}

int main(int argc, char const **argv) {
    // read in the number of possible hidden states and emissions
    int states, emissions, observations_length;
    if (scanf("%d %d", &states, &emissions) != 2) {
#ifdef DEBUG
        fprintf(stderr, "Cannot read # possible states and emissions.\n");
#endif // DEBUG
        exit(EXIT_FAILURE);
    }
    double *init_probabilities = read_init_probabilities(stdin, states);
    double **transition_matrix = read_transition_matrix(stdin, states);
    double **emission_table = read_emission_table(stdin, states, emissions);
    scanf("%d", &observations_length);
    int *observation_table = read_observation(stdin, observations_length);

    for (int i = 0; i < states; i++) {
        init_probabilities[i] = log(init_probabilities[i]);
        for (int j = 0; j < states; j++)
            transition_matrix[i][j] = log(transition_matrix[i][j]);
        for (int j = 0; j < emissions; j++)
            emission_table[i][j] = log(emission_table[i][j]);
    }
    printf("States: %d, Emissions: %d, Observation_length: %d\n", states,
           emissions, observations_length);

#ifdef DEBUG

    printf("[INIT PROBABILITIES]\n");
    for (int i = 0; i < states; i++)
        printf("%.4e ", init_probabilities[i]);
    printf("\n");
    printf("[EMISSION PROBABILITIES]\n");
    for (int i = 0; i < states; i++) {
        for (int j = 0; j < emissions; j++)
            printf("%.4e ", emission_table[i][j]);
        printf("\n");
    }
    printf("[TRANSITION MATRIX]\n");
    for (int i = 0; i < states; i++) {
        for (int j = 0; j < states; j++)
            printf("%.4e ", transition_matrix[i][j]);
        printf("\n");
    }

    printf("[OBSERVATION TABLE]\n");
    for (int i = 0; i < observations_length; i++) {
        printf("%d ", observation_table[i]);
    }
    printf("\n");
#endif // DEBUG
    double **probs_matrix_sequential;

    u_int64_t start = GetTimeStamp();

//    int *optimal_path_sequential = viterbi_sequential(states, emissions,
//                                                      observation_table,
//                                                      observations_length,
//                                                      init_probabilities,
//                                                      transition_matrix,
//                                                      emission_table,
//                                                      &probs_matrix_sequential);
//    printf("SEQUENTIAL Time: %ld\n", GetTimeStamp() - start);

    start = GetTimeStamp();

    int *optimal_path_cuda = viterbi_cuda(states,
                                          emissions,
                                          observations_length,
                                          observation_table,
                                          init_probabilities,
                                          transition_matrix,
                                          emission_table);
    printf("CUDA Time: %ld\n", GetTimeStamp() - start);

//#ifdef DEBUG
//    printf("[ SEQUENTIAL OPTIMAL PATH ]\n");
//    for (int i = 0; i < observations_length; i++) {
//        printf("%3d ", optimal_path_sequential[i]);
//    }
//    putchar('\n');
//
//    printf("[ CUDA OPTIMAL PATH ]\n");
//    for (int i = 0; i < observations_length; i++) {
//        printf("%3d ", optimal_path_cuda[i]);
//    }
//    putchar('\n');

//#endif
//    for (int k = 0; k < observations_length; ++k) {
//        for (int i = 0; i < states; ++i) {
//            if (probs_matrix_cuda[k][i] != probs_matrix_sequential[k][i]) {
//                printf("(%d, %d)[PROBS ERR]\n(%e, %e)\n", k, i,
//                       probs_matrix_cuda[k][i],
//                       probs_matrix_sequential[k][i]);
//            }
//        }
//    }

    free(init_probabilities);
    free_2D_memory(transition_matrix, states);
    free_2D_memory(emission_table, states);
    return 0;
}

/**
 * Allocate and read an array of initial probabilities from a specified file.
 * Calling function must free() memory allocated.
 * @param f File from which to read the probabilities
 * @param states Number of possible states
 * @return Pointer to array of initial probabilities
 */
double *read_init_probabilities(FILE *f, int states) {
    // allocate memory
    double *init_probs = (double *) malloc(states * sizeof(double));
    assert(init_probs);

    // read in values
    for (int i = 0; i < states; i++)
        if (!fscanf(f, "%lf", init_probs + i))
            return NULL;

    return init_probs;
}

/**
 * Allocate and read an array of observations table from a specified file.
 * Calling function must free() memory allocated.
 * @param f File from which to read the observations table
 * @param observations Number of observations table
 * @return Pointer to array of observations table
 */
int *read_observation(FILE *f, int observations) {
    // allocate memory
    int *observations_table = (int *) malloc(observations * sizeof(int));
    assert(observations_table);

    // read in values
    for (int i = 0; i < observations; i++)
        if (!fscanf(f, "%d", observations_table + i))
            return NULL;

    return observations_table;
}

/**
 * Allocate memory and read a transition matrix from a specified file.
 * Calling function must call free_2D_memory() to free memory.
 * @param f File from which to read the transition matrix
 * @param states Number of possible states (matrix width)
 * @return Pointer to transition matrix
 */
double **read_transition_matrix(FILE *f, int states) {
    // allocate memory
    double **transition_matrix = (double **) malloc(states * sizeof(double *));
    assert(transition_matrix);
    for (int i = 0; i < states; i++) {
        transition_matrix[i] = (double *) malloc(states * sizeof(double));
        assert(transition_matrix[i]);
    }

    // read in values
    for (int i = 0; i < states; i++)
        for (int j = 0; j < states; j++)
            if (!fscanf(f, "%lf", transition_matrix[i] + j))
                return NULL;

    return transition_matrix;
}

/**
 * Allocate memory and read an emission probability table from a specified file.
 * Calling function must call free_2D_memory() to free memory.
 * @param f File from which to read the emission probability table
 * @param states Number of possible states (table height)
 * @param emissions Number of possible emissions (table width)
 * @return Pointer to emission probability table
 */
double **read_emission_table(FILE *f, int states, int emissions) {
    // allocate memory
    double **table = (double **) malloc(states * sizeof(double *));
    assert(table);
    for (int i = 0; i < states; i++) {
        table[i] = (double *) malloc(emissions * sizeof(double));
        assert(table[i]);
    }

    // read in values
    for (int i = 0; i < states; i++)
        for (int j = 0; j < emissions; j++)
            if (!fscanf(f, "%lf", table[i] + j))
                return NULL;

    return table;
}

/**
 * Free memory allocated by read_transition_matrix() or read_emission_table().
 * @param table Pointer to table
 * @param states Number of rows
 */
void free_2D_memory(double **table, int rows) {
    if (!table)
        return;
    for (int i = 0; i < rows; i++)
        free(table[i]);
    free(table);
}